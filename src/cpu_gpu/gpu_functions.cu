#include "hip/hip_runtime.h"
#include "magnification_map.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/inner_product.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <hipfft/hipfft.h>

using namespace gerlumph;

// These two prototype functions are used only to compile this source code.
// The 'static' keyword causes their visibility to be limited to the translation unit (this single .cu or .cpp file where they are defined).
static int myfft2d_r2c(int Nx, int Ny, hipfftDoubleReal* data, hipfftDoubleComplex* Fdata);
static int myfft2d_c2r(int Nx, int Ny, hipfftDoubleComplex* Fdata, hipfftDoubleReal* data);


void MagnificationMap::convolve(Kernel* kernel,EffectiveMap* emap){
  hipfftDoubleReal dum1,dum2;
  
  // Check if "kernel", which is a "profile" variable has the same dimension as the map
  
  //Fourier transform map
  hipfftDoubleComplex* Fmap = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,this->data,Fmap);
  //Fourier transform kernel
  hipfftDoubleComplex* Fkernel = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,kernel->data,Fkernel);
  //Multiply kernel and map
  for(long i=0;i<this->Nx*(this->Ny/2+1);i++){
    dum1 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].x - Fmap[i].y*Fkernel[i].y);
    dum2 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].y + Fmap[i].y*Fkernel[i].x);
    Fmap[i].x = dum1;
    Fmap[i].y = dum2;
  }
  //Inverse Fourier transform
  hipfftDoubleReal* cmap = (hipfftDoubleReal*) calloc(this->Nx*this->Ny,sizeof(hipfftDoubleReal));
  myfft2d_c2r(this->Nx,this->Ny,Fmap,cmap);

  free(Fmap);
  free(Fkernel);

  //Normalize convolved map and crop to emap
  double norm = (double) (this->Nx*this->Ny);
  for(int i=0;i<emap->Ny;i++){
    for(int j=0;j<emap->Nx;j++){
      emap->data[i*emap->Nx+j] = (double) (cmap[emap->top*this->Nx+emap->left+i*this->Nx+j]/norm);
    }
  }

  this->convolved = true;
  free(cmap);
}


Mpd MagnificationMap::getFullMpd(){
  Mpd theMpd(0);
  try {
    if( this->convolved ){
      throw "Map is convolved. It has to be in ray counts in order to use this function.";
    }
    thrust::device_vector<int> counts;
    thrust::device_vector<double> bins;
    thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
    thrust::sort(data.begin(),data.end());

    int num_bins = thrust::inner_product(data.begin(),data.end()-1,data.begin()+1,int(1),thrust::plus<int>(),thrust::not_equal_to<double>());
    counts.resize(num_bins);
    bins.resize(num_bins);

    thrust::reduce_by_key(data.begin(),data.end(),thrust::constant_iterator<int>(1),bins.begin(),counts.begin());
    thrust::host_vector<int> hcounts(counts);
    thrust::host_vector<double> hbins(bins);
    
    theMpd.reset(num_bins);
    for(unsigned int i=0;i<hcounts.size();i++){
      theMpd.counts[i] = (double) (hcounts[i])/(double) (this->Nx*this->Ny);
      theMpd.bins[i]   = ((double) (hbins[i]));
    }
  } catch(const char* msg){
    std::cout << msg << std::endl;
  }
  return theMpd;
}


Mpd MagnificationMap::getBinnedMpd(int Nbins){
  // creating bins which are evenly spaced in log space
  double min = 0.02;
  double max = 200;


  double logmin  = log10(min);
  double logmax  = log10(max);
  double logdbin = (logmax-logmin)/Nbins;
  double* bins   = (double*) calloc(Nbins,sizeof(double));
  for(int i=0;i<Nbins;i++){
    bins[i] = pow(10,logmin+(i+1)*logdbin);
  }

  thrust::device_vector<int>    counts(Nbins);
  thrust::device_vector<double> dbins(bins,bins+Nbins);
  thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
  thrust::sort(data.begin(),data.end());

  // For the following lines to work I need to compile using the flag: --expt-extended-lambda
  //  auto getLog10LambdaFunctor = [=]  __device__ (double x) {return log10(x);};
  //  thrust::transform(data.begin(),data.end(),data.begin(),getLog10LambdaFunctor);

  double range[2] = {min,max};
  thrust::device_vector<double> drange(range,range+2);
  thrust::device_vector<int>    dirange(2);
  thrust::lower_bound(data.begin(),data.end(),drange.begin(),drange.end(),dirange.begin());
  thrust::host_vector<int> hirange(dirange);
  //  std::cout << hirange[0] << " " << hirange[1] << std::endl;

  thrust::upper_bound(data.begin() + hirange[0],data.begin() + hirange[1],dbins.begin(),dbins.end(),counts.begin());
  //  thrust::upper_bound(data.begin(),data.end(),dbins.begin(),dbins.end(),counts.begin());
  thrust::adjacent_difference(counts.begin(),counts.end(),counts.begin());
  thrust::host_vector<int>    hcounts(counts);

  Mpd theMpd(hcounts.size());
  for(unsigned int i=0;i<hcounts.size();i++){
    theMpd.counts[i] = (double) (hcounts[i]) /(double) (this->Nx*this->Ny);
    theMpd.bins[i]   = (double) bins[i];
  }
  free(bins);
  return theMpd;
}


int myfft2d_r2c(int Nx,int Ny,hipfftDoubleReal* data,hipfftDoubleComplex* Fdata){
  int result;
  hipfftHandle plan;
  hipfftDoubleReal* data_GPU;
  hipfftDoubleComplex* Fdata_GPU;

  //allocate and transfer memory to the GPU
  hipMalloc( (void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));
  if( hipGetLastError() != hipSuccess ){
    fprintf(stderr, "Cuda error: Failed to allocate data_GPU\n");
    throw std::bad_alloc();
  }
  hipMemcpy( data_GPU, data, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
  hipMalloc( (void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));
  if( hipGetLastError() != hipSuccess ){
    fprintf(stderr, "Cuda error: Failed to allocate data_GPU\n");
    throw std::bad_alloc();
  }

  //do the fourier transform on the GPU
  result = hipfftPlan2d(&plan,Nx,Ny,HIPFFT_D2Z);
  if( result != HIPFFT_SUCCESS ){
    fprintf(stderr, "CUFFT Error: Unable to create plan\n");
    hipFree(data_GPU);
    hipFree(Fdata_GPU);
    throw std::runtime_error("CUFFT Error: Unable to create plan");
  }
  result = hipfftExecD2Z(plan, data_GPU, Fdata_GPU);
  if( result != HIPFFT_SUCCESS ){
    fprintf(stderr, "CUFFT Error: unable to execute plan\n");
    hipFree(data_GPU);
    hipFree(Fdata_GPU);
    hipfftDestroy(plan);
    throw std::runtime_error("CUFFT Error: unable to execute plan");
  }
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);

  //copy back results
  hipMemcpy(Fdata, Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);

  return 0;
}


int myfft2d_c2r(int Nx, int Ny, hipfftDoubleComplex* Fdata, hipfftDoubleReal* data){
  int result;
  hipfftHandle plan;
  hipfftDoubleComplex* Fdata_GPU;
  hipfftDoubleReal* data_GPU;
  
  //allocate and transfer memory
  hipMalloc((void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));
  if( hipGetLastError() != hipSuccess ){
    fprintf(stderr, "Cuda error: Failed to allocate Fdata_GPU\n");
    throw std::bad_alloc();
  }
  hipMemcpy(Fdata_GPU, Fdata, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
  hipMalloc((void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));
  if( hipGetLastError() != hipSuccess ){
    fprintf(stderr, "Cuda error: Failed to allocate Fdata_GPU\n");
    throw std::bad_alloc();
  }

  //do the inverse fourier transform on the GPU
  result = hipfftPlan2d(&plan,Nx,Ny,HIPFFT_Z2D) ;
  if( result != HIPFFT_SUCCESS ){
    fprintf(stderr, "CUFFT Error: Unable to create plan\n");
    hipFree(Fdata_GPU);
    hipFree(data_GPU);
    throw std::runtime_error("CUFFT Error: Unable to create plan");
  }
  result = hipfftExecZ2D(plan, Fdata_GPU, data_GPU);
  if( result != HIPFFT_SUCCESS ){
    fprintf(stderr, "CUFFT Error: unable to execute plan\n");
    hipFree(Fdata_GPU);
    hipFree(data_GPU);
    hipfftDestroy(plan);
    throw std::runtime_error("CUFFT Error: unable to execute plan");
  }
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);
  
  //copy back results
  hipMemcpy(data, data_GPU, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);
  
  return 0;
}
