#include "hip/hip_runtime.h"
#include "map.hpp"


Map::Map(std::string id){
  this->id = id;
  this->convolved = false;
  std::string file;


  // Read map metadata
  file = this->path + this->id + "/mapmeta.dat";
  std::ifstream myfile(file.c_str());
  myfile >> this->avgmu >> this->avgN;
  myfile >> this->Nx;
  myfile >> this->width;
  myfile >> this->k >> this->g >> this->s;
  myfile.close();
  this->Ny = this->Nx;
  this->height = this->width;
  this->pixSizeRein = this->width/this->Nx;


  // Read map data
  file = this->path + this->id + "/map.bin";

  FILE* ptr_myfile = fopen(file.data(),"rb");
  int* imap = (int*) calloc(this->Nx*this->Ny,sizeof(int));
  fread(imap,sizeof(int),this->Nx*this->Ny,ptr_myfile);
  fclose(ptr_myfile);
  
  //int (4 bytes) and hipfftDoubleReal (8 bytes) do not have the same size, so there has to be a type cast
  double factor = fabs(this->avgmu/this->avgN);
  double muth   = fabs( 1.0/(1.0-pow(this->k,2)-pow(this->g,2)) );
  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  for(long i=0;i<this->Nx*this->Ny;i++){
    this->data[i] = (double) imap[i]*factor/muth;
  }
  free(imap);
}


Map::Map(const Map& other){
  this->id     = other.id;
  this->k      = other.k;
  this->g      = other.g;
  this->s      = other.s;
  this->Nx     = other.Nx;
  this->Ny     = other.Ny;
  this->width  = other.width;
  this->height = other.height;
  this->avgmu  = other.avgmu;
  this->avgN   = other.avgN;
  this->pixSizeRein = other.pixSizeRein;
  this->pixSizePhys = other.pixSizePhys; // in units of [10^14 cm]
  this->convolved   = other.convolved;

  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  for(long i=0;i<this->Nx*this->Ny;i++){
    this->data[i] = other.data[i];
  }
}


void Map::convolve(Profile* profile){
  hipfftDoubleReal dum1,dum2;
  
  // Check if "kernel", which is a "profile" variable has the same dimension as the map

  //Fourier transform map
  hipfftDoubleComplex* Fmap = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,this->data,Fmap);
  //Fourier transform kernel
  hipfftDoubleComplex* Fkernel = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,profile->kernel,Fkernel);
  //Multiply kernel and map
  for(long i=0;i<this->Nx*(this->Ny/2+1);i++){
    dum1 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].x - Fmap[i].y*Fkernel[i].y);
    dum2 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].y + Fmap[i].y*Fkernel[i].x);
    Fmap[i].x = dum1;
    Fmap[i].y = dum2;
  }
  //Inverse Fourier transform
  //  cmap = (hipfftDoubleReal*) calloc(this->res*this->res,sizeof(hipfftDoubleReal));
  //  myfft2d_c2r(this->res,this->res,Fmap,cmap);
  myfft2d_c2r(this->Nx,this->Ny,Fmap,this->data);

  free(Fmap);
  free(Fkernel);


  //Normalize convolved map
  double norm = (double) this->Nx*this->Ny;
  for(int i=0;i<this->Ny;i++){
    for(int j=0;j<this->Nx;j++){
      this->data[i*this->Nx+j] /= norm;
    }
  }

  this->convolved = true;
}


Mpd* Map::getFullMpd(){
  if( this->convolved ){
    std::cout << "Map is convolved. Has to be in ray counts." << std::endl;
    return NULL;
  } else {
    thrust::device_vector<double> bins;
    thrust::device_vector<int> counts;
    thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
    thrust::sort(data.begin(),data.end());
    int num_bins = thrust::inner_product(data.begin(),data.end()-1,data.begin()+1,int(1),thrust::plus<int>(),thrust::not_equal_to<double>());
    counts.resize(num_bins);
    bins.resize(num_bins);
    thrust::reduce_by_key(data.begin(),data.end(),thrust::constant_iterator<int>(1),counts.begin(),bins.begin());
    thrust::host_vector<int> hcounts(counts);
    thrust::host_vector<double> hbins(bins);
    
    Mpd* theMpd = new Mpd(hcounts.size());
    for(unsigned int i=0;i<hcounts.size();i++){
      theMpd->counts[i] = (double) hcounts[i]/(this->Nx*this->Ny);
      theMpd->bins[i]   = (double) hbins[i];
    }
    return theMpd; 
  }
}


Mpd* Map::getBinnedMpd(int Nbins){
  // creating bins which are evenly spaced in log space
  double logmin  = log10(0.02);
  double logmax  = log10(200);
  double logdbin = (logmax-logmin)/Nbins;
  double* bins   = (double*) calloc(Nbins,sizeof(double));
  for(int i=0;i<Nbins;i++){
    bins[i] = pow(10,logmin+(i+1)*logdbin);
  }

  thrust::device_vector<int> counts(Nbins);
  thrust::device_vector<double> dbins(bins,bins+Nbins);
  thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
  thrust::sort(data.begin(),data.end());
  thrust::upper_bound(data.begin(),data.end(),dbins.begin(),dbins.end(),counts.begin());
  thrust::adjacent_difference(counts.begin(),counts.end(),counts.begin());
  thrust::host_vector<int> hcounts(counts);

  Mpd* theMpd = new Mpd(hcounts.size());
  for(unsigned int i=0;i<hcounts.size();i++){
    theMpd->counts[i] = (double) hcounts[i]/(this->Nx*this->Ny);
    theMpd->bins[i]   = (double) bins[i];
  }
  return theMpd;
}


int Map::myfft2d_r2c(int Nx,int Ny,hipfftDoubleReal* data,hipfftDoubleComplex* Fdata){
  hipfftHandle plan;
  hipfftDoubleReal* data_GPU;
  hipfftDoubleComplex* Fdata_GPU;

  //allocate and transfer memory to the GPU
  hipMalloc( (void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));
  hipMemcpy( data_GPU, data, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
  hipMalloc( (void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));

  //do the fourier transform on the GPU
  hipfftPlan2d(&plan,Nx,Ny,HIPFFT_D2Z);
  hipfftExecD2Z(plan, data_GPU, Fdata_GPU);
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);

  //copy back results
  hipMemcpy(Fdata, Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);

  return 0;
}


int Map::myfft2d_c2r(int Nx, int Ny, hipfftDoubleComplex* Fdata, hipfftDoubleReal* data){
  hipfftHandle plan;
  hipfftDoubleComplex* Fdata_GPU;
  hipfftDoubleReal* data_GPU;
  
  //allocate and transfer memory
  hipMalloc((void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));
  hipMemcpy(Fdata_GPU, Fdata, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
  hipMalloc((void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));

  //do the inverse fourier transform on the GPU
  hipfftPlan2d(&plan,Nx,Ny,HIPFFT_Z2D) ;
  hipfftExecZ2D(plan, Fdata_GPU, data_GPU);
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);
  
  //copy back results
  hipMemcpy(data, data_GPU, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);
  
  return 0;
}


void Map::writeMapPNG(const std::string filename,int sampling){

  // read, sample, and scale map
  long Ntot = this->Nx*this->Ny/pow(sampling,2);
  int* colors = (int*) calloc(Ntot,sizeof(int));
  this->scaleMap(Ntot,colors,sampling);
  
  // read rgb values from table file (or select them from a stored list of rgb color tables)
  int* rgb = (int*) calloc(3*256,sizeof(int));
  readRGB("rgb.dat",rgb);

  // write image
  writeImage(filename,this->Nx/sampling,this->Ny/sampling,colors,rgb);
}


void Map::scaleMap(int Ntot,int* colors,int sampling){
  double scale_max = 1.6;
  double scale_min = -1.6;
  double scale_fac = 255/(fabs(scale_min) + scale_max);
  double dum,dum2;

  for(long i=0;i<this->Nx*this->Ny;i+=sampling){
    dum = log10(this->data[i]);
    if( dum < scale_min ){
      dum = scale_min;
    }
    if( dum > scale_max ){
      dum = scale_max;
    }
    
    dum2 = (dum + fabs(scale_min))*scale_fac;
    colors[i] = (int) round(dum2);
  }
}


void Map::readRGB(const std::string filename,int* rgb){
  int r,g,b;
  std::ifstream istr(filename);
  
  for(int i=0;i<256;i++){
    istr >> r >> g >> b;
    rgb[i*3] = r;
    rgb[i*3 + 1] = g;
    rgb[i*3 + 2] = b;
  }
}


void Map::writeImage(const std::string fname, int width,int height,int* colors,int* rgb){
  FILE* fp            = fopen(fname.data(), "wb");
  png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  png_infop info_ptr  = png_create_info_struct(png_ptr);
  
  png_init_io(png_ptr, fp);
  png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);
  png_write_info(png_ptr, info_ptr);
  
  
  png_bytep row = (png_bytep) malloc(3 * width * sizeof(png_byte));
  int cindex;
  for(int j=0;j<height;j++) {
    for(int i=0;i<width;i++) {
      cindex = colors[j*width+i];
      
      row[i*3]   = rgb[cindex*3];
      row[i*3+1] = rgb[cindex*3 + 1];
      row[i*3+2] = rgb[cindex*3 + 2];
    }
    png_write_row(png_ptr, row);
  }
  png_write_end(png_ptr, NULL);
  
  
  fclose(fp);
  png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
  png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
  free(row);
}

