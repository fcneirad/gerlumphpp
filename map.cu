#include "hip/hip_runtime.h"
#include "map.hpp"


Map::Map(std::string id){
  this->id = id;
  this->convolved = false;
  std::string file;


  // Read map metadata
  file = this->path + this->id + "/mapmeta.dat";
  std::ifstream myfile(file.c_str());
  myfile >> this->avgmu >> this->avgN;
  myfile >> this->Nx;
  myfile >> this->width;
  myfile >> this->k >> this->g >> this->s;
  myfile.close();
  this->Ny = this->Nx;
  this->height = this->width;
  this->pixSizeRein = this->width/this->Nx;


  // Read map data
  file = this->path + this->id + "/map.bin";

  FILE* ptr_myfile = fopen(file.data(),"rb");
  int* imap = (int*) calloc(this->Nx*this->Ny,sizeof(int));
  fread(imap,sizeof(int),this->Nx*this->Ny,ptr_myfile);
  fclose(ptr_myfile);
  
  //int (4 bytes) and hipfftDoubleReal (8 bytes) do not have the same size, so there has to be a type cast
  double factor = fabs(this->avgmu/this->avgN);
  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  for(long i=0;i<this->Nx*this->Ny;i++){
    this->data[i] = (double) imap[i]*factor;
  }
  free(imap);
}


Map::Map(const Map& other){
  this->id     = other.id;
  this->k      = other.k;
  this->g      = other.g;
  this->s      = other.s;
  this->Nx     = other.Nx;
  this->Ny     = other.Ny;
  this->width  = other.width;
  this->height = other.height;
  this->avgmu  = other.avgmu;
  this->avgN   = other.avgN;
  this->pixSizeRein = other.pixSizeRein;
  this->pixSizePhys = other.pixSizePhys; // in units of [10^14 cm]
  this->convolved   = other.convolved;

  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  for(long i=0;i<this->Nx*this->Ny;i++){
    this->data[i] = other.data[i];
  }
}


void Map::convolve(Profile* profile){
  hipfftDoubleReal dum1,dum2;
  
  // Check if "kernel", which is a "profile" variable has the same dimension as the map

  //Fourier transform map
  hipfftDoubleComplex* Fmap = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,this->data,Fmap);
  //Fourier transform kernel
  hipfftDoubleComplex* Fkernel = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,profile->kernel,Fkernel);
  //Multiply kernel and map
  for(long i=0;i<this->Nx*(this->Ny/2+1);i++){
    dum1 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].x - Fmap[i].y*Fkernel[i].y);
    dum2 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].y + Fmap[i].y*Fkernel[i].x);
    Fmap[i].x = dum1;
    Fmap[i].y = dum2;
  }
  //Inverse Fourier transform
  //  cmap = (hipfftDoubleReal*) calloc(this->res*this->res,sizeof(hipfftDoubleReal));
  //  myfft2d_c2r(this->res,this->res,Fmap,cmap);
  myfft2d_c2r(this->Nx,this->Ny,Fmap,this->data);

  free(Fmap);
  free(Fkernel);


  //Normalize convolved map
  double norm = (double) this->Nx*this->Ny;
  for(int i=0;i<this->Ny;i++){
    for(int j=0;j<this->Nx;j++){
      this->data[i*this->Nx+j] /= norm;
    }
  }

  this->convolved = true;
}


Mpd* Map::getFullMpd(){
  if( this->convolved ){
    std::cout << "Map is convolved. Has to be in ray counts." << std::endl;
    return NULL;
  } else {
    thrust::device_vector<double> bins;
    thrust::device_vector<int> counts;
    thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
    thrust::sort(data.begin(),data.end());
    int num_bins = thrust::inner_product(data.begin(),data.end()-1,data.begin()+1,int(1),thrust::plus<int>(),thrust::not_equal_to<double>());
    counts.resize(num_bins);
    bins.resize(num_bins);
    thrust::reduce_by_key(data.begin(),data.end(),thrust::constant_iterator<int>(1),counts.begin(),bins.begin());
    thrust::host_vector<int> hcounts(counts);
    thrust::host_vector<double> hbins(bins);
    
    Mpd* theMpd = new Mpd(hcounts.size());
    for(unsigned int i=0;i<hcounts.size();i++){
      theMpd->counts[i] = (double) hcounts[i]/(this->Nx*this->Ny);
      theMpd->bins[i]   = (double) hbins[i]*this->avgmu/this->avgN;
    }
    return theMpd; 
  }
}


Mpd* Map::getBinnedMpd(int Nbins){
  double min  = log10(0.02);
  double max  = log10(200);
  double dbin = (max-min)/Nbins;
  double* bins = (double*) calloc(Nbins,sizeof(double));
  for(int i=0;i<Nbins;i++){
    bins[i] = pow(10,min+(i+1)*dbin);
  }

  thrust::device_vector<int> counts(Nbins);
  thrust::device_vector<double> dbins(bins,bins+Nbins);
  thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
  thrust::sort(data.begin(), data.end());
  thrust::upper_bound(data.begin(),data.end(),dbins.begin(),dbins.end(),counts.begin());
  thrust::adjacent_difference(counts.begin(),counts.end(),counts.begin());
  thrust::host_vector<int> hcounts(counts);

  Mpd* theMpd = new Mpd(hcounts.size());
  for(unsigned int i=0;i<hcounts.size();i++){
    theMpd->counts[i] = (double) hcounts[i]/(this->Nx*this->Ny);
    theMpd->bins[i]   = (double) bins[i]*this->avgmu/this->avgN;
  }
  return theMpd;
}


int Map::myfft2d_r2c(int Nx,int Ny,hipfftDoubleReal* data,hipfftDoubleComplex* Fdata){
  hipfftHandle plan;
  hipfftDoubleReal* data_GPU;
  hipfftDoubleComplex* Fdata_GPU;

  //allocate and transfer memory to the GPU
  hipMalloc( (void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));
  hipMemcpy( data_GPU, data, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
  hipMalloc( (void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));

  //do the fourier transform on the GPU
  hipfftPlan2d(&plan,Nx,Ny,HIPFFT_D2Z);
  hipfftExecD2Z(plan, data_GPU, Fdata_GPU);
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);

  //copy back results
  hipMemcpy(Fdata, Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);

  return 0;
}


int Map::myfft2d_c2r(int Nx, int Ny, hipfftDoubleComplex* Fdata, hipfftDoubleReal* data){
  hipfftHandle plan;
  hipfftDoubleComplex* Fdata_GPU;
  hipfftDoubleReal* data_GPU;
  
  //allocate and transfer memory
  hipMalloc((void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));
  hipMemcpy(Fdata_GPU, Fdata, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
  hipMalloc((void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));

  //do the inverse fourier transform on the GPU
  hipfftPlan2d(&plan,Nx,Ny,HIPFFT_Z2D) ;
  hipfftExecZ2D(plan, Fdata_GPU, data_GPU);
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);
  
  //copy back results
  hipMemcpy(data, data_GPU, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);
  
  return 0;
}

