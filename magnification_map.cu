#include "hip/hip_runtime.h"
#include "magnification_map.hpp"


MagnificationMap::MagnificationMap(std::string id,double Rein){
  this->id = id;
  this->imageType = "map";
  this->convolved = false;
  std::string file;


  // Read map metadata
  file = this->path + this->id + "/mapmeta.dat";
  std::ifstream myfile(file.c_str());
  myfile >> this->avgmu >> this->avgN;
  myfile >> this->Nx;
  myfile >> this->width;
  myfile >> this->k >> this->g >> this->s;
  myfile.close();
  this->Ny = this->Nx;
  this->height = this->width;
  this->pixSizePhys = Rein*this->width/this->Nx; // in units of [10^14 cm]


  // Read map data
  file = this->path + this->id + "/map.bin";

  FILE* ptr_myfile = fopen(file.data(),"rb");
  int* imap = (int*) calloc(this->Nx*this->Ny,sizeof(int));
  fread(imap,sizeof(int),this->Nx*this->Ny,ptr_myfile);
  fclose(ptr_myfile);
  
  //int (4 bytes) and hipfftDoubleReal (8 bytes) do not have the same size, so there has to be a type cast
  double factor = fabs(this->avgmu/this->avgN);
  double muth   = fabs( 1.0/(pow(1.0-this->k,2)-pow(this->g,2)) );
  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  for(long i=0;i<this->Nx*this->Ny;i++){
    this->data[i] = (double) (imap[i]*factor/muth);
  }
  free(imap);
}


MagnificationMap::MagnificationMap(const MagnificationMap& other){
  this->imageType = other.imageType;
  this->id     = other.id;
  this->k      = other.k;
  this->g      = other.g;
  this->s      = other.s;
  this->Nx     = other.Nx;
  this->Ny     = other.Ny;
  this->width  = other.width;
  this->height = other.height;
  this->avgmu  = other.avgmu;
  this->avgN   = other.avgN;
  this->pixSizePhys = other.pixSizePhys; // in units of [10^14 cm]
  this->convolved   = other.convolved;

  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  for(long i=0;i<this->Nx*this->Ny;i++){
    this->data[i] = other.data[i];
  }
}


void MagnificationMap::convolve(Kernel* kernel,EffectiveMap* emap){
  hipfftDoubleReal dum1,dum2;
  
  // Check if "kernel", which is a "profile" variable has the same dimension as the map

  //Fourier transform map
  hipfftDoubleComplex* Fmap = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,this->data,Fmap);
  //Fourier transform kernel
  hipfftDoubleComplex* Fkernel = (hipfftDoubleComplex*) calloc(this->Nx*(this->Ny/2+1),sizeof(hipfftDoubleComplex));
  myfft2d_r2c(this->Nx,this->Ny,kernel->data,Fkernel);
  //Multiply kernel and map
  for(long i=0;i<this->Nx*(this->Ny/2+1);i++){
    dum1 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].x - Fmap[i].y*Fkernel[i].y);
    dum2 = (hipfftDoubleReal) (Fmap[i].x*Fkernel[i].y + Fmap[i].y*Fkernel[i].x);
    Fmap[i].x = dum1;
    Fmap[i].y = dum2;
  }
  //Inverse Fourier transform
  hipfftDoubleReal* cmap = (hipfftDoubleReal*) calloc(this->Nx*this->Ny,sizeof(hipfftDoubleReal));
  myfft2d_c2r(this->Nx,this->Ny,Fmap,cmap);

  free(Fmap);
  free(Fkernel);

  //Normalize convolved map and crop to emap
  double norm = (double) (this->Nx*this->Ny);
  for(int i=0;i<emap->Ny;i++){
    for(int j=0;j<emap->Nx;j++){
      emap->data[i*emap->Nx+j] = (double) (cmap[emap->top*this->Nx+emap->left+i*this->Nx+j]/norm);
    }
  }

  this->convolved = true;
  free(cmap);
}


Mpd MagnificationMap::getFullMpd(){
  if( this->convolved ){
    std::cout << "Map is convolved. It has to be in ray counts in order to use this function." << std::endl;
    throw "This is an exception!";
  } else {
    double muth   = fabs(1.0/(pow(1.0-this->k,2)-pow(this->g,2)));

    thrust::device_vector<int> counts;
    thrust::device_vector<double> bins;
    thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
    thrust::sort(data.begin(),data.end());
    
    int num_bins = thrust::inner_product(data.begin(),data.end()-1,data.begin()+1,int(1),thrust::plus<int>(),thrust::not_equal_to<double>());
    counts.resize(num_bins);
    bins.resize(num_bins);
    thrust::reduce_by_key(data.begin(),data.end(),thrust::constant_iterator<int>(1),bins.begin(),counts.begin());
    thrust::host_vector<int> hcounts(counts);
    thrust::host_vector<double> hbins(bins);
    
    Mpd theMpd(hcounts.size());
    for(unsigned int i=0;i<hcounts.size();i++){
      theMpd.counts[i] = (double) (hcounts[i])/(double) (this->Nx*this->Ny);
      theMpd.bins[i]   = ((double) (hbins[i]));
    }
    return theMpd; 
  }
}


Mpd MagnificationMap::getBinnedMpd(int Nbins){
  // creating bins which are evenly spaced in log space
  double min = 0.02;
  double max = 200;


  double logmin  = log10(min);
  double logmax  = log10(max);
  double logdbin = (logmax-logmin)/Nbins;
  double* bins   = (double*) calloc(Nbins,sizeof(double));
  for(int i=0;i<Nbins;i++){
    bins[i] = pow(10,logmin+(i+1)*logdbin);
  }

  thrust::device_vector<int>    counts(Nbins);
  thrust::device_vector<double> dbins(bins,bins+Nbins);
  thrust::device_vector<double> data(this->data,this->data+this->Nx*this->Ny);
  thrust::sort(data.begin(),data.end());

  // For the following lines to work I need to compile using the flag: --expt-extended-lambda
  //  auto getLog10LambdaFunctor = [=]  __device__ (double x) {return log10(x);};
  //  thrust::transform(data.begin(),data.end(),data.begin(),getLog10LambdaFunctor);

  double range[2] = {min,max};
  thrust::device_vector<double> drange(range,range+2);
  thrust::device_vector<int>    dirange(2);
  thrust::lower_bound(data.begin(),data.end(),drange.begin(),drange.end(),dirange.begin());
  thrust::host_vector<int> hirange(dirange);
  //  std::cout << hirange[0] << " " << hirange[1] << std::endl;

  thrust::upper_bound(data.begin() + hirange[0],data.begin() + hirange[1],dbins.begin(),dbins.end(),counts.begin());
  //  thrust::upper_bound(data.begin(),data.end(),dbins.begin(),dbins.end(),counts.begin());
  thrust::adjacent_difference(counts.begin(),counts.end(),counts.begin());
  thrust::host_vector<int>    hcounts(counts);

  Mpd theMpd(hcounts.size());
  for(unsigned int i=0;i<hcounts.size();i++){
    theMpd.counts[i] = (double) (hcounts[i]) /(double) (this->Nx*this->Ny);
    theMpd.bins[i]   = (double) bins[i];
  }
  free(bins);
  return theMpd;
}


int MagnificationMap::myfft2d_r2c(int Nx,int Ny,hipfftDoubleReal* data,hipfftDoubleComplex* Fdata){
  hipfftHandle plan;
  hipfftDoubleReal* data_GPU;
  hipfftDoubleComplex* Fdata_GPU;

  //allocate and transfer memory to the GPU
  hipMalloc( (void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));
  hipMemcpy( data_GPU, data, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
  hipMalloc( (void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));

  //do the fourier transform on the GPU
  hipfftPlan2d(&plan,Nx,Ny,HIPFFT_D2Z);
  hipfftExecD2Z(plan, data_GPU, Fdata_GPU);
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);

  //copy back results
  hipMemcpy(Fdata, Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);

  return 0;
}


int MagnificationMap::myfft2d_c2r(int Nx, int Ny, hipfftDoubleComplex* Fdata, hipfftDoubleReal* data){
  hipfftHandle plan;
  hipfftDoubleComplex* Fdata_GPU;
  hipfftDoubleReal* data_GPU;
  
  //allocate and transfer memory
  hipMalloc((void**) &Fdata_GPU, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex));
  hipMemcpy(Fdata_GPU, Fdata, Nx*(Ny/2+1)*sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
  hipMalloc((void**) &data_GPU, Nx*Ny*sizeof(hipfftDoubleReal));

  //do the inverse fourier transform on the GPU
  hipfftPlan2d(&plan,Nx,Ny,HIPFFT_Z2D) ;
  hipfftExecZ2D(plan, Fdata_GPU, data_GPU);
  //  hipDeviceSynchronize();
  hipDeviceSynchronize();
  hipfftDestroy(plan);
  
  //copy back results
  hipMemcpy(data, data_GPU, Nx*Ny*sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);
  hipFree(data_GPU);
  hipFree(Fdata_GPU);
  
  return 0;
}




EffectiveMap::EffectiveMap(int offset,MagnificationMap* map){
  this->top    = offset;
  this->bottom = offset;
  this->left   = offset;
  this->right  = offset;

  this->imageType = map->imageType;
  this->pixSizePhys = map->pixSizePhys;
  this->Nx = map->Nx - 2*offset;
  this->Ny = map->Ny - 2*offset;
  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  this->width = map->width*this->Nx/map->Nx;
  this->height = map->height*this->Ny/map->Ny;

  this->k = map->k;
  this->g = map->g;
  this->s = map->s;
  this->avgmu = map->avgmu;
  this->avgN = map->avgN;

  this->convolved = true;
}

EffectiveMap::EffectiveMap(int top,int bottom,int left,int right,MagnificationMap* map){
  this->top    = top;
  this->bottom = bottom;
  this->left   = left;
  this->right  = right;

  this->imageType = map->imageType;
  this->pixSizePhys = map->pixSizePhys;
  this->Nx = map->Nx - left - right;
  this->Ny = map->Ny - top - bottom;
  this->data = (double*) calloc(this->Nx*this->Ny,sizeof(double));
  this->width = map->width*this->Nx/map->Nx;
  this->height = map->height*this->Ny/map->Ny;

  this->k = map->k;
  this->g = map->g;
  this->s = map->s;
  this->avgmu = map->avgmu;
  this->avgN = map->avgN;

  this->convolved = true;
}
